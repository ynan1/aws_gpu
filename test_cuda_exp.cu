#include <iostream>
#include <random>
#include <ctime>
#include <thread>
#include <float.h>
#include <algorithm>
#include <numeric>
#include <functional>
#include <atomic>
#include <cmath>
#include <hip/hip_runtime.h>




#define N 1000000000  // Correctly define as integer
#define THREADS_PER_BLOCK 1024


using namespace std;

__device__ float sum_arr= 0.0f; // Initialize sum_arr in device memory

// CUDA kernel
__global__ void max_n(float* max_val, const float* d_in) {
    int idx = threadIdx.x;
    int stride = blockDim.x;
    __shared__ float shared_max[THREADS_PER_BLOCK];
    float max_l = -FLT_MAX;
    //shared_max[idx] = (idx < N) ? d_in[idx] : -FLT_MAX; // Initialize shared memory with input values or -FLT_MAX
    __syncthreads();
    // Find the maximum value per thread block
    for (int i=idx; i < N; i += stride) {
        max_l = fmaxf(max_l, d_in[i]);
    }
    shared_max[idx] = max_l;
    __syncthreads();
    // Reduce to find the maximum value in shared memory
   for (int s = stride / 2; s > 0; s >>= 1) {
       if (idx < s && idx + s < N) {
           shared_max[idx] = fmaxf(shared_max[idx], shared_max[idx + s]);
       }
       __syncthreads();
   }
    // Write the maximum value to global memory
    if (idx == 0) {
        *max_val = shared_max[0];
    }
}

__global__ void exponent(float* d_out, const float* d_in,float* max_val) {
    //2D Block indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        d_out[idx] = expf(d_in[idx] - *max_val);
    }
    __syncthreads();

    //Add all the values in d_out
    atomicAdd(&sum_arr, d_out[idx]);
    __syncthreads();

    // Normalize the output
    if (idx < N) {
        d_out[idx] /= sum_arr;
    }
}

// Thread-safe random number generator
thread_local std::mt19937 generator(std::random_device{}());

// Generate random numbers for a part of the array
void drand(float* arr, int size) {
    uniform_real_distribution<float> distribution(0.0f, 3.0f);
    for (int i = 0; i < size; i++) {
        arr[i] = distribution(generator);
    }
}

int main(int argc,char* argv[]) {
    #ifdef USE_MANAGED
    float* din= nullptr;
    float* dout= nullptr;
    float* max_val= nullptr; // Initialize max_val to a very small value
    cudaMallocManaged(&max_val, sizeof(float));
    cudaMallocManaged(&din, sizeof(float) * N);
    cudaMallocManaged(&dout, sizeof(float) * N);
    #else
    float* din = new float[N];
    float* dout = new float[N];

    if (!din || !dout) {
        cerr << "Host memory allocation failed!" << endl;
        return -1;
    }

    #endif

    // Fill input array using 4 CPU threads
    thread threads[4];
    int chunk = N / 4;
    for (int i = 0; i < 4; ++i) {
        threads[i] = thread(drand, din + i * chunk, chunk);
    }
    for (int i = 0; i < 4; ++i) {
        threads[i].join();
    }

    float max_val_cpu=*max_element(din,din+N);

    // CUDA device memory
#ifdef USE_MANAGED
    cudaMemPrefetchAsync(din, sizeof(float) * N, 0);
#else
    float *gpu_din, *gpu_dout, *max_val_ptr;
    hipError_t err;
    err = hipMalloc(&gpu_din, sizeof(float) * N);
    err = hipMalloc(&gpu_dout, sizeof(float) * N);
    err = hipMalloc(&max_val_ptr, sizeof(float));

    if (err != hipSuccess) {
        cerr << "GPU memory allocation failed!" << endl;
        return -1;
    }

    hipMemcpy(gpu_din, din, sizeof(float) * N, hipMemcpyHostToDevice);
    //cudaMemcpy(max_val_ptr, &max_val, sizeof(float), cudaMemcpyHostToDevice);
#endif

    // Use streams if enabled
#ifdef USE_STREAM
    const int nStreams = 3;
    cudaStream_t stream[nStreams];
    int streamSize = N / nStreams;

    for (int i = 0; i < nStreams; ++i)
        cudaStreamCreate(&stream[i]);

    for (int i = 0; i < nStreams; ++i) {
        int offset = i * streamSize;
        cudaMemcpyAsync(gpu_din + offset, din + offset, streamSize * sizeof(float), cudaMemcpyHostToDevice, stream[i]);
        int blocks = (streamSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        exponent<<<blocks, THREADS_PER_BLOCK, 0, stream[i]>>>(gpu_dout + offset, gpu_din + offset);
        cudaMemcpyAsync(dout + offset, gpu_dout + offset, streamSize * sizeof(float), cudaMemcpyDeviceToHost, stream[i]);
    }

    for (int i = 0; i < nStreams; ++i)
        cudaStreamSynchronize(stream[i]);

#else
int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
#ifdef USE_MANAGED
    // Launch the kernel (single stream)
    // Find the max_val of the array
    max_n<<<1, THREADS_PER_BLOCK>>>(max_val, din);
    exponent<<<blocks, THREADS_PER_BLOCK>>>(dout, din, max_val);
    cudaDeviceSynchronize();
    cout<<*max_val<<" "<<max_val_cpu<<endl;
#else
    // Launch the kernel (single stream)
    max_n<<<1,THREADS_PER_BLOCK>>>(max_val_ptr,gpu_din);
    exponent<<<blocks, THREADS_PER_BLOCK>>>(gpu_dout, gpu_din,max_val_ptr);
    hipMemcpy(dout, gpu_dout, sizeof(float) * N, hipMemcpyDeviceToHost);
    //cudaMemcpy(&max_val, max_val_ptr, sizeof(float), cudaMemcpyDeviceToHost);
    //cout<<max_val<<" "<<max_val_cpu<<endl;
#endif
#endif

    float cum_abs_err=0.0f;
	float max_abs=FLT_MIN;
    float sum_arr_host=0.0f;
    float* exp_arr=new float[N];

    for (int i = 0; i < N; ++i) {
        exp_arr[i] = expf(din[i] - max_val_cpu);
        sum_arr_host += exp_arr[i];
    }

	for (int i=0;i<N;i++){
        float exp= exp_arr[i]/sum_arr_host;
		cum_abs_err+=fabs(dout[i]-exp);
		max_abs=fmax(max_abs,fabs(dout[i]-exp));
	}

	cout<<"cumm_abs_error: "<<cum_abs_err<<endl;
	cout<<"max_abs_err: "<<max_abs<<endl;    


// Cleanup
    delete[] exp_arr;

#ifdef USE_MANAGED
    cudaFree(din);
    cudaFree(dout);
    cudaFree(max_val);
#else
    delete[] din;
    delete[] dout;
    hipFree(gpu_din);
    hipFree(gpu_dout);
    hipFree(max_val_ptr);
#endif


#ifdef USE_STREAM
    for (int i = 0; i < nStreams; ++i)
        cudaStreamDestroy(stream[i]);
#endif

    return 0;
}

